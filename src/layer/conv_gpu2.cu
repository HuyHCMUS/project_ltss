#include "hip/hip_runtime.h"
#include "conv_gpu2.h"
#include <math.h>
#include <iostream>

void ConvGPU2::init() {
  height_out = (1 + (height_in - height_kernel + 2 * pad_h) / stride);
  width_out =   (1 + (width_in - width_kernel + 2 * pad_w) / stride);
  dim_out = height_out * width_out * channel_out;

  weight.resize(channel_in * height_kernel * width_kernel, channel_out);
  bias.resize(channel_out);
  grad_weight.resize(channel_in * height_kernel * width_kernel, channel_out);
  grad_bias.resize(channel_out);
  set_normal_random(weight.data(), weight.size(), 0, 0.01);
  set_normal_random(bias.data(), bias.size(), 0, 0.01);
  //std::cout << weight.colwise().sum() << std::endl;
  //std::cout << weight.colwise().sum() + bias.transpose() << std::endl;
}

// im2col, used for bottom
// image size: Vector (height_in * width_in * channel_in)
// data_col size: Matrix (hw_out, hw_kernel * channel_in)
void ConvGPU2::im2col(const Vector& image, Matrix& data_col) {
  int hw_in = height_in * width_in;
  int hw_kernel = height_kernel * width_kernel;
  int hw_out = height_out * width_out;
  // im2col
  data_col.resize(hw_out, hw_kernel * channel_in);
  for (int c = 0; c < channel_in; c ++) {
    Vector map = image.block(hw_in * c, 0, hw_in, 1);  // c-th channel map
    for (int i = 0; i < hw_out; i ++) {
      int step_h = i / width_out;
      int step_w = i % width_out;
      int start_idx = step_h * width_in * stride + step_w * stride;  // left-top idx of window
      for (int j = 0; j < hw_kernel; j ++) {
        int cur_col = start_idx % width_in + j % width_kernel - pad_w;  // col after padding
        int cur_row = start_idx / width_in + j / width_kernel - pad_h;
        if (cur_col < 0 || cur_col >= width_in || cur_row < 0 ||
            cur_row >= height_in) {
          data_col(i, c * hw_kernel + j) = 0;
        }
        else {
          //int pick_idx = start_idx + (j / width_kernel) * width_in + j % width_kernel;
          int pick_idx = cur_row * width_in + cur_col;
          data_col(i, c * hw_kernel + j) = map(pick_idx);  // pick which pixel
        }
      }
    }
  }
}

__global__ void im2col_kernel(const float* image, float* data_col, int height_in, int width_in, 
                            int channel_in, int height_out, int width_out, 
                            int height_kernel, int width_kernel, int stride, int pad_h, int pad_w) 
{
  int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

  int hw_kernel = height_kernel * width_kernel;
  int hw_out = height_out * width_out;

  int step_h = global_idx / width_out;
  int step_w = global_idx % width_out;
  int start_idx = step_h * width_in * stride + step_w * stride;

  for (int c = 0; c < channel_in; c ++) {
    for (int j = 0; j < hw_kernel; j ++) {
      int cur_col = start_idx % width_in + j % width_kernel - pad_w; 
      int cur_row = start_idx / width_in + j / width_kernel - pad_h;

      if (cur_col < 0 || cur_col >= width_in || cur_row < 0 || cur_row >= height_in) {
        //data_col[global_idx * channel_in * hw_kernel +  c * hw_kernel + j] = 0;
        data_col[(c * hw_kernel + j) * hw_out + global_idx] = 0;
      }
      else {
        int pick_idx = cur_row * width_in + cur_col;
        data_col[(c * hw_kernel + j) * hw_out + global_idx] = image[pick_idx + c * height_in * width_in];
      }
    }
  }
}


void ConvGPU2::forward(const Matrix& bottom) {
  int n_sample = bottom.cols();
  top.resize(height_out * width_out * channel_out, n_sample);
  data_cols.resize(n_sample);
  for (int i = 0; i < n_sample; i ++) {
    // im2col
    Matrix data_col;
    data_col.resize(height_out * width_out, height_kernel * width_kernel * channel_in);

    float* d_image;
    float* d_data_col;

    CHECK(hipMalloc(&d_image, bottom.col(i).size() * sizeof(float)));
    CHECK(hipMalloc(&d_data_col, data_col.size() * sizeof(float)));

    CHECK(hipMemcpy(d_image, bottom.col(i).data(), bottom.col(i).size() * sizeof(float), hipMemcpyHostToDevice));

    dim3 blockSize(32);
    dim3 gridSize((height_out * width_out - 1) / blockSize.x + 1); 

    im2col_kernel<<<gridSize, blockSize>>>(d_image, d_data_col, height_in, width_in, channel_in, height_out, width_out, height_kernel, width_kernel, stride, pad_h, pad_w);
    
    hipMemcpy(data_col.data(), d_data_col, data_col.size() * sizeof(float), hipMemcpyDeviceToHost);

    CHECK(hipFree(d_image));
    CHECK(hipFree(d_data_col));
    //im2col(bottom.col(i), data_col);
    data_cols[i] = data_col;
    // conv by product
    Matrix result = data_col * weight;  // result: (hw_out, channel_out)
    result.rowwise() += bias.transpose();
    top.col(i) = Eigen::Map<Vector>(result.data(), result.size());
  }
}

// col2im, used for grad_bottom
// data_col size: Matrix (hw_out, hw_kernel * channel_in)
// image size: Vector (height_in * width_in * channel_in)
void ConvGPU2::col2im(const Matrix& data_col, Vector& image) {
  int hw_in = height_in * width_in;
  int hw_kernel = height_kernel * width_kernel;
  int hw_out = height_out * width_out;
  // col2im
  image.resize(hw_in * channel_in);
  image.setZero();
  for (int c = 0; c < channel_in; c ++) {
    for (int i = 0; i < hw_out; i ++) {
      int step_h = i / width_out;
      int step_w = i % width_out;
      int start_idx = step_h * width_in * stride + step_w * stride;  // left-top idx of window
      for (int j = 0; j < hw_kernel; j ++) {
        int cur_col = start_idx % width_in + j % width_kernel - pad_w;  // col after padding
        int cur_row = start_idx / width_in + j / width_kernel - pad_h;
        if (cur_col < 0 || cur_col >= width_in || cur_row < 0 ||
            cur_row >= height_in) {
          continue;
        }
        else {
          //int pick_idx = start_idx + (j / width_kernel) * width_in + j % width_kernel;
          int pick_idx = cur_row * width_in + cur_col;
          image(c * hw_in + pick_idx) += data_col(i, c * hw_kernel + j);  // pick which pixel
        }
      }
    }
  }
}

void ConvGPU2::backward(const Matrix& bottom, const Matrix& grad_top) {
  int n_sample = bottom.cols();
  grad_weight.setZero();
  grad_bias.setZero();
  grad_bottom.resize(height_in * width_in * channel_in, n_sample);
  grad_bottom.setZero();
  for (int i = 0; i < n_sample; i ++) {
    // im2col of grad_top
    Matrix grad_top_i = grad_top.col(i);
    Matrix grad_top_i_col = Eigen::Map<Matrix>(grad_top_i.data(),
                              height_out * width_out, channel_out);
    // d(L)/d(w) = \sum{ d(L)/d(z_i) * d(z_i)/d(w) }
    grad_weight += data_cols[i].transpose() * grad_top_i_col;
    // d(L)/d(b) = \sum{ d(L)/d(z_i) * d(z_i)/d(b) }
    grad_bias += grad_top_i_col.colwise().sum().transpose();
    // d(L)/d(x) = \sum{ d(L)/d(z_i) * d(z_i)/d(x) } = d(L)/d(z)_col * w'
    Matrix grad_bottom_i_col = grad_top_i_col * weight.transpose();
    // col2im of grad_bottom
    Vector grad_bottom_i;
    col2im(grad_bottom_i_col, grad_bottom_i);
    grad_bottom.col(i) = grad_bottom_i;
  }
}

void ConvGPU2::update(Optimizer& opt) {
  Vector::AlignedMapType weight_vec(weight.data(), weight.size());
  Vector::AlignedMapType bias_vec(bias.data(), bias.size());
  Vector::ConstAlignedMapType grad_weight_vec(grad_weight.data(), grad_weight.size());
  Vector::ConstAlignedMapType grad_bias_vec(grad_bias.data(), grad_bias.size());

  opt.update(weight_vec, grad_weight_vec);
  opt.update(bias_vec, grad_bias_vec);
}

std::vector<float> ConvGPU2::get_parameters() const {
  std::vector<float> res(weight.size() + bias.size());
  // Copy the data of weights and bias to a long vector
  std::copy(weight.data(), weight.data() + weight.size(), res.begin());
  std::copy(bias.data(), bias.data() + bias.size(), res.begin() + weight.size());
  return res;
}

void ConvGPU2::set_parameters(const std::vector<float>& param) {
  if(static_cast<int>(param.size()) != weight.size() + bias.size())
      throw std::invalid_argument("Parameter size does not match");
  std::copy(param.begin(), param.begin() + weight.size(), weight.data());
  std::copy(param.begin() + weight.size(), param.end(), bias.data());
}

std::vector<float> ConvGPU2::get_derivatives() const {
  std::vector<float> res(grad_weight.size() + grad_bias.size());
  // Copy the data of weights and bias to a long vector
  std::copy(grad_weight.data(), grad_weight.data() + grad_weight.size(), res.begin());
  std::copy(grad_bias.data(), grad_bias.data() + grad_bias.size(),
            res.begin() + grad_weight.size());
  return res;
}
